#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <fstream>
#include <vector>
#include <iomanip>
#include <time.h>

#define AES_BLOCK_SIZE 16
#define THREADS_PER_BLOCK 128
#define NUM_ROUNDS 10

// AES S-Box
__device__ const unsigned char sbox[256] = {
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

// Round constant array
__device__ const unsigned char Rcon[10] = {
    0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1B, 0x36
};

// SubBytes step of AES encryption
__device__ void SubBytes(unsigned char* state) {
    for (int i = 0; i < AES_BLOCK_SIZE; ++i) {
        state[i] = sbox[state[i]];
    }
}

// ShiftRows step of AES encryption
__device__ void ShiftRows(unsigned char* state) {
    uint8_t temp = state[1 * 4 + 0];
    state[1 * 4 + 0] = state[1 * 4 + 1];
    state[1 * 4 + 1] = state[1 * 4 + 2];
    state[1 * 4 + 2] = state[1 * 4 + 3];
    state[1 * 4 + 3] = temp;

    //                           2            
    temp = state[2 * 4 + 0];
    state[2 * 4 + 0] = state[2 * 4 + 2];
    state[2 * 4 + 2] = temp;
    temp = state[2 * 4 + 1];
    state[2 * 4 + 1] = state[2 * 4 + 3];
    state[2 * 4 + 3] = temp;

    //                              3            
    temp = state[3 * 4 + 3];
    state[3 * 4 + 3] = state[3 * 4 + 2];
    state[3 * 4 + 2] = state[3 * 4 + 1];
    state[3 * 4 + 1] = state[3 * 4 + 0];
    state[3 * 4 + 0] = temp;
}

__device__ uint8_t multiply(uint8_t a, uint8_t b) {
    uint8_t result = 0;
    uint8_t high_bit_set;
    for (int i = 0; i < 8; ++i) {
        if (b & 1) {
            result ^= a;
        }
        high_bit_set = a & 0x80;
        a <<= 1;
        if (high_bit_set) {
            a ^= 0x1B; //                         x^8 + x^4 + x^3 + x + 1
        }
        b >>= 1;
    }
    return result;
}
// MixColumns step of AES encryption
__device__ void MixColumns(unsigned char* state) {
    //printf("AAAAAAAAAAAAAAAAAA\n");
    for (size_t i = 0; i < 16; ++i) {
        //printf("%02x ", state[i]);
    }
    //printf("\n");
    //printf("\nAAAAAAAAAAAAAAAAAA\n");
    for (int i = 0; i < 4; ++i) {
        uint8_t a = state[i];
        //printf("%02x ", state[i]);
        uint8_t b = state[4 + i];
        //printf("%02x ", state[i+4]);
        uint8_t c = state[8 + i];
        //printf("%02x ", state[i + 8]);
        uint8_t d = state[12 + i];
        //printf("%02x ", state[i + 12]);
        //printf("%02x %02x %02x %02x\n", a, b, c, d);

        state[i] = (unsigned char)(multiply(a, 0x02) ^ multiply(b, 0x03) ^ c ^ d);
        state[4 + i] = (unsigned char)(a ^ multiply(b, 0x02) ^ multiply(c, 0x03) ^ d);
        state[8 + i] = (unsigned char)(a ^ b ^ multiply(c, 0x02) ^ multiply(d, 0x03));
        state[12 + i] = (unsigned char)(multiply(a, 0x03) ^ b ^ c ^ multiply(d, 0x02));
        uint8_t e = multiply(a, 0x02) ^ multiply(b, 0x03) ^ c ^ d;
        uint8_t f = a ^ multiply(b, 0x02) ^ multiply(c, 0x03) ^ d;
        uint8_t g = a ^ b ^ multiply(c, 0x02) ^ multiply(d, 0x03);
        uint8_t h = multiply(a, 0x03) ^ b ^ c ^ multiply(d, 0x02);
        //printf("%02x %02x %02x %02x\n", e, f, g, h);
    }
    //printf("\nAAAAAAAAAAAAAAAAAA\n");
}

// AddRoundKey step of AES encryption
__device__ void AddRoundKey(unsigned char* state, const unsigned char* roundKey, int mode) {
    //printf("\n---------\n");
    if (mode == 1) {
        for (int i = 0; i < 4; ++i) {
            //printf("%02x ", roundKey[i]);
            //printf();
            for (int j = 0; j < 4; ++j) {
                state[i * 4 + j] ^= roundKey[i + 4 * j];
                //printf("%02x ", roundKey[i*4+j]);
            }
        }
    }
    else {
        for (int i = 0; i < 4; ++i) {
            //printf("%02x ", roundKey[i]);
            //printf();
            for (int j = 0; j < 4; ++j) {
                state[i * 4 + j] ^= roundKey[i * 4 + j];
            }
        }
    }
    //printf("\n---------\n");
}

// KeyExpansion for AES encryption
__device__ void SubWord(unsigned char* word) {
    for (int i = 0; i < 4; ++i) {
        word[i] = sbox[word[i]];
    }
}

__device__ void RotWord(unsigned char* word) {
    unsigned char temp = word[0];
    word[0] = word[1];
    word[1] = word[2];
    word[2] = word[3];
    word[3] = temp;
}

__device__ void XorWords(unsigned char* word1, const unsigned char* word2) {
    for (int i = 0; i < 4; ++i) {
        word1[i] ^= word2[i];
    }
}

__device__ void KeyExpansion(const unsigned char* key, unsigned char* roundKeys) {
    // Copy the original key to the first round key
    for (int i = 0; i < AES_BLOCK_SIZE; ++i) {
        roundKeys[i] = key[i];
    }

    int roundConstantIndex = 0;
    int currentIndex = AES_BLOCK_SIZE;

    while (currentIndex < AES_BLOCK_SIZE * (NUM_ROUNDS + 1)) {
        unsigned char tempWord[4];
        for (int i = 0; i < 4; ++i) {
            tempWord[i] = roundKeys[currentIndex - 4 + i];
        }

        if (currentIndex % AES_BLOCK_SIZE == 0) {
            RotWord(tempWord);
            SubWord(tempWord);
            tempWord[0] ^= Rcon[roundConstantIndex++];
        }

        XorWords(tempWord, &roundKeys[currentIndex - AES_BLOCK_SIZE]);

        for (int i = 0; i < 4; ++i) {
            roundKeys[currentIndex++] = tempWord[i];
        }
    }
}

__device__ void IncrementCounter(char* counter) {
    //            128-                
    for (int i = 15; i >= 0; --i) {
        if (counter[i] == 0xFF) {
            counter[i] = 0x00;
        }
        else {
            counter[i]++;
            break;
        }
    }
}

// AES encryption kernel for CTR mode
__global__ void aesCtrEncryptKernel(const unsigned char* plaintext, const unsigned char* key,
    const unsigned char* nonce, const int numBlocks, unsigned char* ciphertext) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int blockIndex = idx; blockIndex < numBlocks; blockIndex += gridDim.x * blockDim.x) {
        // Calculate the counter value (nonce + block index)
        char counter[AES_BLOCK_SIZE];
        for (int i = 0; i < AES_BLOCK_SIZE; ++i) {
            counter[i] = 0x00;
        }
        for (int k = 0; k < 1; k++) {
            IncrementCounter(counter);
        }

        // Generate round keys
        unsigned char roundKeys[176];
        KeyExpansion(key, roundKeys);

        // Encrypt the counter using AES
        unsigned char state[AES_BLOCK_SIZE];
        for (int i = 0; i < AES_BLOCK_SIZE; ++i) {
            state[i] = counter[i];
        }
        AddRoundKey(state, &roundKeys[0 * AES_BLOCK_SIZE], 1);

        for (int round = 0; round < 10; ++round) {
            SubBytes(state);

            ShiftRows(state);

            if (round < 9) {
                MixColumns(state);

            }
            if (round == 0) {
                AddRoundKey(state, &roundKeys[(round + 1) * AES_BLOCK_SIZE], 1);
            }
            else {
                AddRoundKey(state, &roundKeys[(round + 1) * AES_BLOCK_SIZE], 1);
            }

        }

        // XOR the plaintext block with the encrypted counter to get the ciphertext block
        for (int k = 0; k < 1; k++) {
            for (int i = 0; i < 4; ++i) {
                for (int j = 0; j < 4; ++j) {
                    ciphertext[blockIndex * AES_BLOCK_SIZE + i + j * 4] = state[i * 4 + j] ^ plaintext[blockIndex * AES_BLOCK_SIZE + i + j * 4];
                    //ciphertext[blockIndex * AES_BLOCK_SIZE + i + j * 4] = '\0';
                    printf("%d - %d - %02x\n", blockIndex * AES_BLOCK_SIZE + i + j * 4, blockIndex, state[i * 4 + j] ^ plaintext[blockIndex * AES_BLOCK_SIZE + i + j * 4]);
                    //printf("%02x\n", ciphertext[blockIndex * AES_BLOCK_SIZE + i + j * 4]);
                }


            }
        }
    }
}

// Wrapper function for AES encryption in CTR mode
void aesCtrEncrypt(const uint8_t* plaintext, const char* key, const char* nonce,
    const int numBlocksCUDA, unsigned char* ciphertext) {
    clock_t start_time = clock();

    // Allocate device memory
    unsigned char* d_plaintext, * d_key, * d_nonce, * d_ciphertext;
    hipMalloc((void**)&d_plaintext, numBlocksCUDA * AES_BLOCK_SIZE);
    hipMalloc((void**)&d_key, AES_BLOCK_SIZE);
    hipMalloc((void**)&d_nonce, AES_BLOCK_SIZE);
    hipMalloc((void**)&d_ciphertext, numBlocksCUDA * AES_BLOCK_SIZE);
    clock_t test1_time = clock();
    double elapsed_time = ((double)(test1_time - start_time) / CLOCKS_PER_SEC) * 1000;
    //printf("\nПрограмма выполнилась за: %.2f миллисекунд\n", elapsed_time);
    // Copy data from host to device
    hipMemcpyAsync(d_plaintext, plaintext, numBlocksCUDA * AES_BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_key, key, AES_BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_nonce, nonce, AES_BLOCK_SIZE, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    clock_t test2_time = clock();
    elapsed_time = ((double)(test2_time - start_time) / CLOCKS_PER_SEC) * 1000;
    //printf("\nПрограмма выполнилась за: %.2f миллисекунд\n", elapsed_time);
    // Launch the CUDA kernel
    //unsigned int num_states = size_bytes >> 4;
    int numThreads = THREADS_PER_BLOCK;
    int numBlocks = (numBlocksCUDA + numThreads - 1) / numThreads;
    //int blocksPerGrid = (num_states + threadsPerBlock - 1) / threadsPerBlock;
    aesCtrEncryptKernel << <numBlocks, numThreads >> > (d_plaintext, d_key, d_nonce, numBlocksCUDA, d_ciphertext);
    clock_t test3_time = clock();
    elapsed_time = ((double)(test3_time - start_time) / CLOCKS_PER_SEC) * 1000;
    //printf("ANSWER:\n");
    //printf("\nПрограмма выполнилась за: %.2f миллисекунд\n", elapsed_time);
    //std::vector<uint8_t> ciphertext(AES_BLOCK_SIZE * numBlocks);
    //unsigned char* h_ciphertext = new unsigned char[AES_BLOCK_SIZE * numBlocks];
    hipMemcpyAsync(ciphertext, d_ciphertext, AES_BLOCK_SIZE * numBlocks, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // Write ciphertext to a file

    // Copy the result back to the host
    //hipMemcpyAsync(cipherFinal.data(), d_ciphertext, numBlocksCUDA * AES_BLOCK_SIZE, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    // Free device memory
    hipFree(d_plaintext);
    //hipFree(d_key);
    //hipFree(d_nonce);
    hipFree(d_ciphertext);
    clock_t test4_time = clock();
    elapsed_time = ((double)(test4_time - start_time) / CLOCKS_PER_SEC) * 1000;
    //printf("\nПрограмма выполнилась за: %.2f миллисекунд\n", elapsed_time);
}

int main() {
    // Example usage
    //uint8_t plaintext[] = "abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789abcdef123456789xabcdef123456789";
    //clock_t start_time = clock();
    //unsigned char ciphertext[AES_BLOCK_SIZE * numBlocks];
    std::ifstream plaintextFile("plaintext.txt", std::ios::binary);
    if (!plaintextFile.is_open()) {
        std::cerr << "Error opening plaintext file.\n";
        return 1;
    }

    plaintextFile.seekg(0, std::ios::end);
    size_t plaintextSize = plaintextFile.tellg();
    plaintextFile.seekg(0, std::ios::beg);

    std::vector<uint8_t> plaintext(plaintextSize);
    plaintextFile.read(reinterpret_cast<char*>(plaintext.data()), plaintextSize);
    plaintextFile.close();
    //plaintext[15] = '0';
    //plaintext[31] = '0';

    char key[] = "kkkkeeeeyyyy....";
    char nonce[] = "abcdefgh12345678";

    int numBlocks = (plaintextSize + AES_BLOCK_SIZE - 1) / AES_BLOCK_SIZE;
    //unsigned char ciphertext[AES_BLOCK_SIZE * numBlocks];
    unsigned char* ciphertext = new unsigned char[AES_BLOCK_SIZE * numBlocks];
    //hipMalloc((void**)&d_ciphertext, AES_BLOCK_SIZE * numBlocks);
    //std::vector<uint8_t> ciphertext(AES_BLOCK_SIZE * numBlocks);
    clock_t start_time = clock();
    aesCtrEncrypt(plaintext.data(), key, nonce, numBlocks, ciphertext);
    clock_t end_time = clock();
    std::cout << "Encrypted Message: ";
    for (int i = 0; i < 16; ++i) {
        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)ciphertext[i];
    }
    std::cout << std::endl;
    int length = sizeof(plaintext) - 1;
    // Display the encrypted message

    //clock_t end_time = clock();

    // Вычисляем время в миллисекундах
    double elapsed_time = ((double)(end_time - start_time) / CLOCKS_PER_SEC) * 1000;

    // Выводим результат
    printf("\nПрограмма выполнилась за: %.2f миллисекунд\n", elapsed_time);



    return 0;
}
